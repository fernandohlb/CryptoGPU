
/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <stdbool.h>
/****************************** MACROS ******************************/
// Obtain bit "b" from the left and shift it "c" places from the right
#define BITNUM(a,b,c) (((a[(b)/8] >> (7 - (b%8))) & 0x01) << (c))
#define BITNUMINTR(a,b,c) ((((a) >> (31 - (b))) & 0x00000001) << (c))
#define BITNUMINTL(a,b,c) ((((a) << (b)) & 0x80000000) >> (c))

// This macro converts a 6 bit block with the S-Box row defined as the first and last
// bits to a 6 bit block with the row defined by the first two bits.
#define SBOXBIT(a) (((a) & 0x20) | (((a) & 0x1f) >> 1) | (((a) & 0x01) << 4))

#define DES_BLOCK_SIZE 8                // DES operates on 8 bytes at a time


/**************************** DATA TYPES ****************************/
typedef unsigned char BYTE;             // 8-bit byte
typedef unsigned int  WORD;             // 32-bit word, change to "long" for 16-bit machines

/**************************** VARIABLES *****************************/
/*static const BYTE sbox1[64] = {
    14,  4,  13,  1,   2, 15,  11,  8,   3, 10,   6, 12,   5,  9,   0,  7,
     0, 15,   7,  4,  14,  2,  13,  1,  10,  6,  12, 11,   9,  5,   3,  8,
     4,  1,  14,  8,  13,  6,   2, 11,  15, 12,   9,  7,   3, 10,   5,  0,
    15, 12,   8,  2,   4,  9,   1,  7,   5, 11,   3, 14,  10,  0,   6, 13
};
static const BYTE sbox2[64] = {
    15,  1,   8, 14,   6, 11,   3,  4,   9,  7,   2, 13,  12,  0,   5, 10,
     3, 13,   4,  7,  15,  2,   8, 14,  12,  0,   1, 10,   6,  9,  11,  5,
     0, 14,   7, 11,  10,  4,  13,  1,   5,  8,  12,  6,   9,  3,   2, 15,
    13,  8,  10,  1,   3, 15,   4,  2,  11,  6,   7, 12,   0,  5,  14,  9
};
static const BYTE sbox3[64] = {
    10,  0,   9, 14,   6,  3,  15,  5,   1, 13,  12,  7,  11,  4,   2,  8,
    13,  7,   0,  9,   3,  4,   6, 10,   2,  8,   5, 14,  12, 11,  15,  1,
    13,  6,   4,  9,   8, 15,   3,  0,  11,  1,   2, 12,   5, 10,  14,  7,
     1, 10,  13,  0,   6,  9,   8,  7,   4, 15,  14,  3,  11,  5,   2, 12
};
static const BYTE sbox4[64] = {
     7, 13,  14,  3,   0,  6,   9, 10,   1,  2,   8,  5,  11, 12,   4, 15,
    13,  8,  11,  5,   6, 15,   0,  3,   4,  7,   2, 12,   1, 10,  14,  9,
    10,  6,   9,  0,  12, 11,   7, 13,  15,  1,   3, 14,   5,  2,   8,  4,
     3, 15,   0,  6,  10,  1,  13,  8,   9,  4,   5, 11,  12,  7,   2, 14
};
static const BYTE sbox5[64] = {
     2, 12,   4,  1,   7, 10,  11,  6,   8,  5,   3, 15,  13,  0,  14,  9,
    14, 11,   2, 12,   4,  7,  13,  1,   5,  0,  15, 10,   3,  9,   8,  6,
     4,  2,   1, 11,  10, 13,   7,  8,  15,  9,  12,  5,   6,  3,   0, 14,
    11,  8,  12,  7,   1, 14,   2, 13,   6, 15,   0,  9,  10,  4,   5,  3
};
static const BYTE sbox6[64] = {
    12,  1,  10, 15,   9,  2,   6,  8,   0, 13,   3,  4,  14,  7,   5, 11,
    10, 15,   4,  2,   7, 12,   9,  5,   6,  1,  13, 14,   0, 11,   3,  8,
     9, 14,  15,  5,   2,  8,  12,  3,   7,  0,   4, 10,   1, 13,  11,  6,
     4,  3,   2, 12,   9,  5,  15, 10,  11, 14,   1,  7,   6,  0,   8, 13
};
static const BYTE sbox7[64] = {
     4, 11,   2, 14,  15,  0,   8, 13,   3, 12,   9,  7,   5, 10,   6,  1,
    13,  0,  11,  7,   4,  9,   1, 10,  14,  3,   5, 12,   2, 15,   8,  6,
     1,  4,  11, 13,  12,  3,   7, 14,  10, 15,   6,  8,   0,  5,   9,  2,
     6, 11,  13,  8,   1,  4,  10,  7,   9,  5,   0, 15,  14,  2,   3, 12
};
static const BYTE sbox8[64] = {
    13,  2,   8,  4,   6, 15,  11,  1,  10,  9,   3, 14,   5,  0,  12,  7,
     1, 15,  13,  8,  10,  3,   7,  4,  12,  5,   6, 11,   0, 14,   9,  2,
     7, 11,   4,  1,   9, 12,  14,  2,   0,  6,  10, 13,  15,  3,   5,  8,
     2,  1,  14,  7,   4, 10,   8, 13,  15, 12,   9,  0,   3,  5,   6, 11
};
*/
/*********************** FUNCTION DEFINITIONS ***********************/
// Initial (Inv)Permutation step
__host__ __device__ void IP(WORD state[], const BYTE in[])
{
    state[0] = BITNUM(in,57,31) | BITNUM(in,49,30) | BITNUM(in,41,29) | BITNUM(in,33,28) |
                  BITNUM(in,25,27) | BITNUM(in,17,26) | BITNUM(in,9,25) | BITNUM(in,1,24) |
                  BITNUM(in,59,23) | BITNUM(in,51,22) | BITNUM(in,43,21) | BITNUM(in,35,20) |
                  BITNUM(in,27,19) | BITNUM(in,19,18) | BITNUM(in,11,17) | BITNUM(in,3,16) |
                  BITNUM(in,61,15) | BITNUM(in,53,14) | BITNUM(in,45,13) | BITNUM(in,37,12) |
                  BITNUM(in,29,11) | BITNUM(in,21,10) | BITNUM(in,13,9) | BITNUM(in,5,8) |
                  BITNUM(in,63,7) | BITNUM(in,55,6) | BITNUM(in,47,5) | BITNUM(in,39,4) |
                  BITNUM(in,31,3) | BITNUM(in,23,2) | BITNUM(in,15,1) | BITNUM(in,7,0);

    state[1] = BITNUM(in,56,31) | BITNUM(in,48,30) | BITNUM(in,40,29) | BITNUM(in,32,28) |
                  BITNUM(in,24,27) | BITNUM(in,16,26) | BITNUM(in,8,25) | BITNUM(in,0,24) |
                  BITNUM(in,58,23) | BITNUM(in,50,22) | BITNUM(in,42,21) | BITNUM(in,34,20) |
                  BITNUM(in,26,19) | BITNUM(in,18,18) | BITNUM(in,10,17) | BITNUM(in,2,16) |
                  BITNUM(in,60,15) | BITNUM(in,52,14) | BITNUM(in,44,13) | BITNUM(in,36,12) |
                  BITNUM(in,28,11) | BITNUM(in,20,10) | BITNUM(in,12,9) | BITNUM(in,4,8) |
                  BITNUM(in,62,7) | BITNUM(in,54,6) | BITNUM(in,46,5) | BITNUM(in,38,4) |
                  BITNUM(in,30,3) | BITNUM(in,22,2) | BITNUM(in,14,1) | BITNUM(in,6,0);
}

__host__ __device__ void InvIP(WORD state[], BYTE in[])
{
    in[0] = BITNUMINTR(state[1],7,7) | BITNUMINTR(state[0],7,6) | BITNUMINTR(state[1],15,5) |
              BITNUMINTR(state[0],15,4) | BITNUMINTR(state[1],23,3) | BITNUMINTR(state[0],23,2) |
              BITNUMINTR(state[1],31,1) | BITNUMINTR(state[0],31,0);

    in[1] = BITNUMINTR(state[1],6,7) | BITNUMINTR(state[0],6,6) | BITNUMINTR(state[1],14,5) |
              BITNUMINTR(state[0],14,4) | BITNUMINTR(state[1],22,3) | BITNUMINTR(state[0],22,2) |
              BITNUMINTR(state[1],30,1) | BITNUMINTR(state[0],30,0);

    in[2] = BITNUMINTR(state[1],5,7) | BITNUMINTR(state[0],5,6) | BITNUMINTR(state[1],13,5) |
              BITNUMINTR(state[0],13,4) | BITNUMINTR(state[1],21,3) | BITNUMINTR(state[0],21,2) |
              BITNUMINTR(state[1],29,1) | BITNUMINTR(state[0],29,0);

    in[3] = BITNUMINTR(state[1],4,7) | BITNUMINTR(state[0],4,6) | BITNUMINTR(state[1],12,5) |
              BITNUMINTR(state[0],12,4) | BITNUMINTR(state[1],20,3) | BITNUMINTR(state[0],20,2) |
              BITNUMINTR(state[1],28,1) | BITNUMINTR(state[0],28,0);

    in[4] = BITNUMINTR(state[1],3,7) | BITNUMINTR(state[0],3,6) | BITNUMINTR(state[1],11,5) |
              BITNUMINTR(state[0],11,4) | BITNUMINTR(state[1],19,3) | BITNUMINTR(state[0],19,2) |
              BITNUMINTR(state[1],27,1) | BITNUMINTR(state[0],27,0);

    in[5] = BITNUMINTR(state[1],2,7) | BITNUMINTR(state[0],2,6) | BITNUMINTR(state[1],10,5) |
              BITNUMINTR(state[0],10,4) | BITNUMINTR(state[1],18,3) | BITNUMINTR(state[0],18,2) |
              BITNUMINTR(state[1],26,1) | BITNUMINTR(state[0],26,0);

    in[6] = BITNUMINTR(state[1],1,7) | BITNUMINTR(state[0],1,6) | BITNUMINTR(state[1],9,5) |
              BITNUMINTR(state[0],9,4) | BITNUMINTR(state[1],17,3) | BITNUMINTR(state[0],17,2) |
              BITNUMINTR(state[1],25,1) | BITNUMINTR(state[0],25,0);

    in[7] = BITNUMINTR(state[1],0,7) | BITNUMINTR(state[0],0,6) | BITNUMINTR(state[1],8,5) |
              BITNUMINTR(state[0],8,4) | BITNUMINTR(state[1],16,3) | BITNUMINTR(state[0],16,2) |
              BITNUMINTR(state[1],24,1) | BITNUMINTR(state[0],24,0);
}

__host__ __device__ WORD f(WORD state, const BYTE key[])
{

	const BYTE sbox1[64] = {
	    14,  4,  13,  1,   2, 15,  11,  8,   3, 10,   6, 12,   5,  9,   0,  7,
	     0, 15,   7,  4,  14,  2,  13,  1,  10,  6,  12, 11,   9,  5,   3,  8,
	     4,  1,  14,  8,  13,  6,   2, 11,  15, 12,   9,  7,   3, 10,   5,  0,
	    15, 12,   8,  2,   4,  9,   1,  7,   5, 11,   3, 14,  10,  0,   6, 13
	};
	const BYTE sbox2[64] = {
	    15,  1,   8, 14,   6, 11,   3,  4,   9,  7,   2, 13,  12,  0,   5, 10,
	     3, 13,   4,  7,  15,  2,   8, 14,  12,  0,   1, 10,   6,  9,  11,  5,
	     0, 14,   7, 11,  10,  4,  13,  1,   5,  8,  12,  6,   9,  3,   2, 15,
	    13,  8,  10,  1,   3, 15,   4,  2,  11,  6,   7, 12,   0,  5,  14,  9
	};
	const BYTE sbox3[64] = {
	    10,  0,   9, 14,   6,  3,  15,  5,   1, 13,  12,  7,  11,  4,   2,  8,
	    13,  7,   0,  9,   3,  4,   6, 10,   2,  8,   5, 14,  12, 11,  15,  1,
	    13,  6,   4,  9,   8, 15,   3,  0,  11,  1,   2, 12,   5, 10,  14,  7,
	     1, 10,  13,  0,   6,  9,   8,  7,   4, 15,  14,  3,  11,  5,   2, 12
	};
	const BYTE sbox4[64] = {
	     7, 13,  14,  3,   0,  6,   9, 10,   1,  2,   8,  5,  11, 12,   4, 15,
	    13,  8,  11,  5,   6, 15,   0,  3,   4,  7,   2, 12,   1, 10,  14,  9,
	    10,  6,   9,  0,  12, 11,   7, 13,  15,  1,   3, 14,   5,  2,   8,  4,
	     3, 15,   0,  6,  10,  1,  13,  8,   9,  4,   5, 11,  12,  7,   2, 14
	};
	const BYTE sbox5[64] = {
	     2, 12,   4,  1,   7, 10,  11,  6,   8,  5,   3, 15,  13,  0,  14,  9,
	    14, 11,   2, 12,   4,  7,  13,  1,   5,  0,  15, 10,   3,  9,   8,  6,
	     4,  2,   1, 11,  10, 13,   7,  8,  15,  9,  12,  5,   6,  3,   0, 14,
	    11,  8,  12,  7,   1, 14,   2, 13,   6, 15,   0,  9,  10,  4,   5,  3
	};
	const BYTE sbox6[64] = {
	    12,  1,  10, 15,   9,  2,   6,  8,   0, 13,   3,  4,  14,  7,   5, 11,
	    10, 15,   4,  2,   7, 12,   9,  5,   6,  1,  13, 14,   0, 11,   3,  8,
	     9, 14,  15,  5,   2,  8,  12,  3,   7,  0,   4, 10,   1, 13,  11,  6,
	     4,  3,   2, 12,   9,  5,  15, 10,  11, 14,   1,  7,   6,  0,   8, 13
	};
	const BYTE sbox7[64] = {
	     4, 11,   2, 14,  15,  0,   8, 13,   3, 12,   9,  7,   5, 10,   6,  1,
	    13,  0,  11,  7,   4,  9,   1, 10,  14,  3,   5, 12,   2, 15,   8,  6,
	     1,  4,  11, 13,  12,  3,   7, 14,  10, 15,   6,  8,   0,  5,   9,  2,
	     6, 11,  13,  8,   1,  4,  10,  7,   9,  5,   0, 15,  14,  2,   3, 12
	};
	const BYTE sbox8[64] = {
	    13,  2,   8,  4,   6, 15,  11,  1,  10,  9,   3, 14,   5,  0,  12,  7,
	     1, 15,  13,  8,  10,  3,   7,  4,  12,  5,   6, 11,   0, 14,   9,  2,
	     7, 11,   4,  1,   9, 12,  14,  2,   0,  6,  10, 13,  15,  3,   5,  8,
	     2,  1,  14,  7,   4, 10,   8, 13,  15, 12,   9,  0,   3,  5,   6, 11
	};
    BYTE lrgstate[6]; //,i;
    WORD t1,t2;

    // Expantion Permutation
    t1 = BITNUMINTL(state,31,0) | ((state & 0xf0000000) >> 1) | BITNUMINTL(state,4,5) |
          BITNUMINTL(state,3,6) | ((state & 0x0f000000) >> 3) | BITNUMINTL(state,8,11) |
          BITNUMINTL(state,7,12) | ((state & 0x00f00000) >> 5) | BITNUMINTL(state,12,17) |
          BITNUMINTL(state,11,18) | ((state & 0x000f0000) >> 7) | BITNUMINTL(state,16,23);

    t2 = BITNUMINTL(state,15,0) | ((state & 0x0000f000) << 15) | BITNUMINTL(state,20,5) |
          BITNUMINTL(state,19,6) | ((state & 0x00000f00) << 13) | BITNUMINTL(state,24,11) |
          BITNUMINTL(state,23,12) | ((state & 0x000000f0) << 11) | BITNUMINTL(state,28,17) |
          BITNUMINTL(state,27,18) | ((state & 0x0000000f) << 9) | BITNUMINTL(state,0,23);

    lrgstate[0] = (t1 >> 24) & 0x000000ff;
    lrgstate[1] = (t1 >> 16) & 0x000000ff;
    lrgstate[2] = (t1 >> 8) & 0x000000ff;
    lrgstate[3] = (t2 >> 24) & 0x000000ff;
    lrgstate[4] = (t2 >> 16) & 0x000000ff;
    lrgstate[5] = (t2 >> 8) & 0x000000ff;

    // Key XOR
    lrgstate[0] ^= key[0];
    lrgstate[1] ^= key[1];
    lrgstate[2] ^= key[2];
    lrgstate[3] ^= key[3];
    lrgstate[4] ^= key[4];
    lrgstate[5] ^= key[5];

    // S-Box Permutation
    state = (sbox1[SBOXBIT(lrgstate[0] >> 2)] << 28) |
              (sbox2[SBOXBIT(((lrgstate[0] & 0x03) << 4) | (lrgstate[1] >> 4))] << 24) |
              (sbox3[SBOXBIT(((lrgstate[1] & 0x0f) << 2) | (lrgstate[2] >> 6))] << 20) |
              (sbox4[SBOXBIT(lrgstate[2] & 0x3f)] << 16) |
              (sbox5[SBOXBIT(lrgstate[3] >> 2)] << 12) |
              (sbox6[SBOXBIT(((lrgstate[3] & 0x03) << 4) | (lrgstate[4] >> 4))] << 8) |
              (sbox7[SBOXBIT(((lrgstate[4] & 0x0f) << 2) | (lrgstate[5] >> 6))] << 4) |
                sbox8[SBOXBIT(lrgstate[5] & 0x3f)];

    // P-Box Permutation
    state = BITNUMINTL(state,15,0) | BITNUMINTL(state,6,1) | BITNUMINTL(state,19,2) |
              BITNUMINTL(state,20,3) | BITNUMINTL(state,28,4) | BITNUMINTL(state,11,5) |
              BITNUMINTL(state,27,6) | BITNUMINTL(state,16,7) | BITNUMINTL(state,0,8) |
              BITNUMINTL(state,14,9) | BITNUMINTL(state,22,10) | BITNUMINTL(state,25,11) |
              BITNUMINTL(state,4,12) | BITNUMINTL(state,17,13) | BITNUMINTL(state,30,14) |
              BITNUMINTL(state,9,15) | BITNUMINTL(state,1,16) | BITNUMINTL(state,7,17) |
              BITNUMINTL(state,23,18) | BITNUMINTL(state,13,19) | BITNUMINTL(state,31,20) |
              BITNUMINTL(state,26,21) | BITNUMINTL(state,2,22) | BITNUMINTL(state,8,23) |
              BITNUMINTL(state,18,24) | BITNUMINTL(state,12,25) | BITNUMINTL(state,29,26) |
              BITNUMINTL(state,5,27) | BITNUMINTL(state,21,28) | BITNUMINTL(state,10,29) |
              BITNUMINTL(state,3,30) | BITNUMINTL(state,24,31);

    // Return the final state value
    return(state);
}

__host__ __device__ void des_key_setup(const BYTE key[], BYTE schedule[][6], bool mode)
{
    WORD i, j, to_gen, C, D;
    const WORD key_rnd_shift[16] = {1,1,2,2,2,2,2,2,1,2,2,2,2,2,2,1};
    const WORD key_perm_c[28] = {56,48,40,32,24,16,8,0,57,49,41,33,25,17,
                                 9,1,58,50,42,34,26,18,10,2,59,51,43,35};
    const WORD key_perm_d[28] = {62,54,46,38,30,22,14,6,61,53,45,37,29,21,
                                 13,5,60,52,44,36,28,20,12,4,27,19,11,3};
    const WORD key_compression[48] = {13,16,10,23,0,4,2,27,14,5,20,9,
                                      22,18,11,3,25,7,15,6,26,19,12,1,
                                      40,51,30,36,46,54,29,39,50,44,32,47,
                                      43,48,38,55,33,52,45,41,49,35,28,31};

    // Permutated Choice #1 (copy the key in, ignoring parity bits).
    for (i = 0, j = 31, C = 0; i < 28; ++i, --j)
        C |= BITNUM(key,key_perm_c[i],j);
    for (i = 0, j = 31, D = 0; i < 28; ++i, --j)
        D |= BITNUM(key,key_perm_d[i],j);

    // Generate the 16 subkeys.
    for (i = 0; i < 16; ++i) {
        C = ((C << key_rnd_shift[i]) | (C >> (28-key_rnd_shift[i]))) & 0xfffffff0;
        D = ((D << key_rnd_shift[i]) | (D >> (28-key_rnd_shift[i]))) & 0xfffffff0;

        // Decryption subkeys are reverse order of encryption subkeys so
        // generate them in reverse if the key schedule is for decryption useage.
        if (mode == false)
            to_gen = 15 - i;
        else /*(if mode == DES_ENCRYPT)*/
            to_gen = i;
        // Initialize the array
        for (j = 0; j < 6; ++j)
            schedule[to_gen][j] = 0;
        for (j = 0; j < 24; ++j)
            schedule[to_gen][j/8] |= BITNUMINTR(C,key_compression[j],7 - (j%8));
        for ( ; j < 48; ++j)
            schedule[to_gen][j/8] |= BITNUMINTR(D,key_compression[j] - 28,7 - (j%8));
    }
}

__host__ __device__ void des_crypt(const BYTE in[], BYTE out[], const BYTE key[][6])
{
    WORD state[2],idx,t;

    IP(state,in);

    for (idx=0; idx < 15; ++idx) {
        t = state[1];
        state[1] = f(state[1],key[idx]) ^ state[0];
        state[0] = t;
    }
    // Perform the final loop manually as it doesn't switch sides
    state[0] = f(state[1],key[15]) ^ state[0];

    InvIP(state,out);
}

__host__ __device__ void three_des_key_setup(const BYTE key[], BYTE schedule[][16][6], bool mode)
{
    if (mode == true) {
        des_key_setup(&key[0],schedule[0],mode);
        des_key_setup(&key[8],schedule[1],!mode);
        des_key_setup(&key[16],schedule[2],mode);
    }
    else /*if (mode == DES_DECRYPT*/ {
        des_key_setup(&key[16],schedule[0],mode);
        des_key_setup(&key[8],schedule[1],!mode);
        des_key_setup(&key[0],schedule[2],mode);
    }
}

__host__ __device__ void three_des_crypt(const BYTE in[], BYTE out[], const BYTE key[][16][6])
{
    des_crypt(in,out,key[0]);
    des_crypt(out,out,key[1]);
    des_crypt(out,out,key[2]);
}


/*********************** FUNCTION DEFINITIONS FOR TEST***********************/
int des_test()
{
    BYTE pt1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xE7};
    BYTE pt2[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
    BYTE pt3[DES_BLOCK_SIZE] = {0x54,0x68,0x65,0x20,0x71,0x75,0x66,0x63};
    BYTE ct1[DES_BLOCK_SIZE] = {0xc9,0x57,0x44,0x25,0x6a,0x5e,0xd3,0x1d};
    BYTE ct2[DES_BLOCK_SIZE] = {0x85,0xe8,0x13,0x54,0x0f,0x0a,0xb4,0x05};
    BYTE ct3[DES_BLOCK_SIZE] = {0xc9,0x57,0x44,0x25,0x6a,0x5e,0xd3,0x1d};
    BYTE ct4[DES_BLOCK_SIZE] = {0xA8,0x26,0xFD,0x8C,0xE5,0x3B,0x85,0x5F};
    BYTE key1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
    BYTE key2[DES_BLOCK_SIZE] = {0x13,0x34,0x57,0x79,0x9B,0xBC,0xDF,0xF1};
    BYTE three_key1[DES_BLOCK_SIZE * 3] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF,
                                           0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF,
                                           0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
    BYTE three_key2[DES_BLOCK_SIZE * 3] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF,
                                           0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF,0x01,
                                           0x45,0x67,0x89,0xAB,0xCD,0xEF,0x01,0x23};

    BYTE schedule[16][6];
    BYTE three_schedule[3][16][6];
    BYTE buf[DES_BLOCK_SIZE];
    int pass = 1;

    des_key_setup(key1, schedule, true);
    des_crypt(pt1, buf, schedule);
    pass = pass && !memcmp(ct1, buf, DES_BLOCK_SIZE);

    des_key_setup(key1, schedule, false);
    des_crypt(ct1, buf, schedule);
    pass = pass && !memcmp(pt1, buf, DES_BLOCK_SIZE);

    des_key_setup(key2, schedule, true);
    des_crypt(pt2, buf, schedule);
    pass = pass && !memcmp(ct2, buf, DES_BLOCK_SIZE);

    des_key_setup(key2, schedule, false);
    des_crypt(ct2, buf, schedule);
    pass = pass && !memcmp(pt2, buf, DES_BLOCK_SIZE);

    three_des_key_setup(three_key1, three_schedule, true);
    three_des_crypt(pt1, buf, three_schedule);
    pass = pass && !memcmp(ct3, buf, DES_BLOCK_SIZE);

    three_des_key_setup(three_key1, three_schedule, false);
    three_des_crypt(ct3, buf, three_schedule);
    pass = pass && !memcmp(pt1, buf, DES_BLOCK_SIZE);

    three_des_key_setup(three_key2, three_schedule, true);
    three_des_crypt(pt3, buf, three_schedule);
    pass = pass && !memcmp(ct4, buf, DES_BLOCK_SIZE);

    three_des_key_setup(three_key2, three_schedule, false);
    three_des_crypt(ct4, buf, three_schedule);
    pass = pass && !memcmp(pt3, buf, DES_BLOCK_SIZE);

    return(pass);
}


__global__ void paralell_enc_dec(char4 *data)
{
	printf("OK");
}

/*********************** CUDA KERNEL FOR ENC DEC ***********************/
__global__ void paralell_enc_dec(const BYTE *data, BYTE *encrypted_data, BYTE *decrypted_data, long size)
{

    BYTE data_buf[DES_BLOCK_SIZE];
    BYTE data_enc[DES_BLOCK_SIZE];
    BYTE data_dec[DES_BLOCK_SIZE];
    BYTE key1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
    BYTE schedule[16][6];

    int i = (blockDim.x * blockIdx.x + threadIdx.x)*DES_BLOCK_SIZE;
   if (i < size){
        for(int j = 0; j < DES_BLOCK_SIZE; j++){	    
            if(i < size){
                data_buf[j] = data[i];
                i++;
            };
        };

        des_key_setup(key1, schedule, true);
        des_crypt(data_buf, data_enc, schedule);

        des_key_setup(key1, schedule, false);
        des_crypt(data_enc, data_dec, schedule);

        i -= DES_BLOCK_SIZE;
        for(int k = 0; k < DES_BLOCK_SIZE; k++){	    
            if(i < size){
                encrypted_data[i] = data_enc[k];
                decrypted_data[i] = data_dec[k];
                i++;
            };
        };
    };
    



}


/*********************** ENC DEC FUNCTION FOR TEST***********************/
void enc_dec_file()
{
    BYTE *h_data;
    BYTE *d_data = NULL;
    BYTE *h_encrypted_data;
    BYTE *h_decrypted_data;
    BYTE *d_encrypted_data;
    BYTE *d_decrypted_data;
    const char *filename = "sample_files/hubble_1.tif";


    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    struct stat st;

    if (stat(filename, &st) == 0){
        h_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
    };

    printf("SIZE of BYTE: %d \n",sizeof(BYTE));
    FILE *file = fopen(filename, "rb");

    if(h_data != NULL && file){
        int current_byte = 0;

        while(fread(&h_data[current_byte], sizeof(BYTE), 1, file) == 1){
            current_byte += 1;
        };
    };

printf("ST.SIZE: %d \n",sizeof(BYTE) *st.st_size);
    h_encrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
    h_decrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
    
    //Alocando o BYTE data no device
    err = hipMalloc((void **)&d_data, sizeof(BYTE) * st.st_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Alocando o BYTE d_encrypted_data no device
    err = hipMalloc((void **)&d_encrypted_data, sizeof(BYTE) * st.st_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device encrypted_data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    //Alocando o BYTE d_decrypted_data no device
    err = hipMalloc((void **)&d_decrypted_data, sizeof(BYTE) * st.st_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device decrypted_data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copy the host BYTE data in host memory to the device in
    // device memory
    printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_data, h_data, sizeof(BYTE) * st.st_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy BYTE data from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the paralell_enc_dec CUDA Kernel
    int threadsPerBlock = 512;
    int blocksPerGrid =( (sizeof(BYTE) * st.st_size)/(threadsPerBlock * DES_BLOCK_SIZE) )+1;
    //int blocksPerGrid = 7853;
    //printf("Tamanho do Arquivo: %d \n", sizeof(BYTE) * st.st_size);
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    paralell_enc_dec<<<blocksPerGrid, threadsPerBlock>>>(d_data,d_encrypted_data,d_decrypted_data,st.st_size);


    err = hipGetLastError();

    if (err != hipSuccess)
    {
	fprintf(stderr, "Failed to launch paralell_enc_dec kernel (error code %s)!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }

    // Copy the device result encrypted_data in device memory to the host result vector
    // in host memory.
    printf("Copy output encrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_encrypted_data, d_encrypted_data, st.st_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector encrypted_data from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result decrypted_data in device memory to the host result vector
    // in host memory.
    printf("Copy output decrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_decrypted_data, d_decrypted_data, sizeof(BYTE) * st.st_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector decrypted_data from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    /*BYTE data_buf[DES_BLOCK_SIZE];
    BYTE data_enc[DES_BLOCK_SIZE];
    BYTE data_dec[DES_BLOCK_SIZE];
    BYTE key1[DES_BLOCK_SIZE] = {0x01,0x23,0x45,0x67,0x89,0xAB,0xCD,0xEF};
    BYTE schedule[16][6];

    for(int i = 0; i < st.st_size; i++){
    //for(int i = 0; i < 32; i++){
        for(int j = 0; j < DES_BLOCK_SIZE; j++){	    
            if(i < st.st_size){
                data_buf[j] = h_data[i];
                i++;
            };
        };

        des_key_setup(key1, schedule, true);
        des_crypt(data_buf, data_enc, schedule);

        des_key_setup(key1, schedule, false);
        des_crypt(data_enc, data_dec, schedule);

        i -= DES_BLOCK_SIZE;
        for(int k = 0; k < DES_BLOCK_SIZE; k++){	    
            if(i < st.st_size){
                h_encrypted_data[i] = data_enc[k];
                h_decrypted_data[i] = data_dec[k];
                i++;
            };
        };

        i--;
	
    };*/

    FILE *enc_file = fopen("hubble_1_enc.tif", "wb+");
    FILE *dec_file = fopen("hubble_1_dec.tif", "wb+");

    fwrite(h_encrypted_data, sizeof(BYTE) * st.st_size, 1, enc_file);
    fwrite(h_decrypted_data, sizeof(BYTE) * st.st_size, 1, dec_file);

    fclose(enc_file);
    fclose(dec_file);

    // Free device global memory
    err = hipFree(d_data);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Data d_data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_encrypted_data);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Data d_encrypted_data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_decrypted_data);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Data d_decrypted_data (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_data);
    free(h_encrypted_data);
    free(h_decrypted_data);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling cudaDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
};


int main()
{
    enc_dec_file();
    printf("DES test: %s\n", des_test() ? "SUCCEEDED" : "FAILED");
    return(0);
}
