#include "aes.h"
#include "../../include/time_utils.h"

int aes_ecb_test(char * path_to_file)
{
    struct timespec wc_start[STEPS_SIZE], wc_end[STEPS_SIZE];
    double cpu_start[STEPS_SIZE], cpu_end[STEPS_SIZE];
    start_timers(cpu_start, wc_start, alloc);
    BYTE *h_data;
    BYTE *d_data = NULL;
    BYTE *h_encrypted_data;
    BYTE *h_decrypted_data;
    BYTE *d_encrypted_data;
    BYTE *d_decrypted_data;
    BYTE key[32] = {0x60,0x3d,0xeb,0x10,0x15,0xca,0x71,0xbe,0x2b,0x73,0xae,0xf0,0x85,0x7d,0x77,0x81,0x1f,0x35,0x2c,0x07,0x3b,0x61,0x08,0xd7,0x2d,0x98,0x10,0xa3,0x09,0x14,0xdf,0xf4};
    WORD h_key_schedule[60];
    WORD * d_key_schedule;
    const char *filename = path_to_file;

    //Key Setup
    aes_key_setup(key, h_key_schedule, 256);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    struct stat st;

    if (stat(filename, &st) != 0){
        printf("Could not find File");
    };

    unsigned long plaintext_size = 1;
    if(st.st_size % 16 != 0){
        plaintext_size = st.st_size + (16 - st.st_size % 16);
    } else {
        plaintext_size = st.st_size;
    }
    printf("Tamanho do Arquivo: %i \n", st.st_size);

    h_data = (BYTE *) malloc(sizeof(BYTE) * plaintext_size);

    FILE *file = fopen(filename, "rb");

    if(h_data != NULL && file){
        unsigned long current_byte = 0;

        while(fread(&h_data[current_byte], sizeof(BYTE), 1, file) == 1){
            current_byte += 1;
        };

        while(current_byte < plaintext_size - 1){
            current_byte += 1;
            h_data[current_byte] = 0;
        }
    };

    h_encrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
    h_decrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
    
    //Alocando o BYTE data no device
    err = hipMalloc((void **)&d_data, sizeof(BYTE) * st.st_size);
    check_cuda_error("Failed to allocate device data", err);    

    //Alocando o BYTE d_encrypted_data no device
    err = hipMalloc((void **)&d_encrypted_data, sizeof(BYTE) * st.st_size);
    check_cuda_error("Failed to allocate device encrypted_data", err);

    //Alocando o BYTE d_decrypted_data no device
    err = hipMalloc((void **)&d_decrypted_data, sizeof(BYTE) * st.st_size);
    check_cuda_error("Failed to allocate device decrypted_data", err);

    //Alocando o WORD key no device
    err = hipMalloc((void **)&d_key_schedule, sizeof(WORD) * 60);
    check_cuda_error("Failed to allocate device data", err);    

    // Copy the host BYTE data in host memory to the device in
    // device memory
    // printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_data, h_data, sizeof(BYTE) * st.st_size, hipMemcpyHostToDevice);
    check_cuda_error("Failed to copy BYTE data from host to device", err);

    // Copy the host WORD key_schedule in host memory to the device in
    // device memory
    // printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_key_schedule, h_key_schedule, sizeof(WORD) * 60, hipMemcpyHostToDevice);
    check_cuda_error("Failed to copy BYTE data from host to device", err);
    end_timers(cpu_end, wc_end, alloc);

    // Launch the aes_encrypt CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =((sizeof(BYTE) * st.st_size)/AES_BLOCK_SIZE/threadsPerBlock)+1;
   
    //Launch encryption
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    start_timers(cpu_start, wc_start, calc);
    aes_encrypt_ecb<<<blocksPerGrid, threadsPerBlock>>>(d_data, st.st_size, d_encrypted_data, d_key_schedule, 256);
    err = hipGetLastError();
    check_cuda_error("Failed to launch aes_encrypt kernel", err);

    // Copy the device result encrypted_data in device memory to the host result vector
    // in host memory.
    // printf("Copy output encrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_encrypted_data, d_encrypted_data, sizeof(BYTE) * st.st_size, hipMemcpyDeviceToHost);
    check_cuda_error("Failed to copy vector encrypted_data from device to host", err);

    //Launch decryption
    // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    aes_decrypt_ecb<<<blocksPerGrid, threadsPerBlock>>>(d_encrypted_data, st.st_size, d_decrypted_data, d_key_schedule, 256);
    err = hipGetLastError();
    check_cuda_error("Failed to launch aes_decrypt kernel", err);
    hipDeviceSynchronize();
    end_timers(cpu_end, wc_end, calc);

    // Copy the device result decrypted_data in device memory to the host result vector
    // in host memory.
    // printf("Copy output decrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_decrypted_data, d_decrypted_data, sizeof(BYTE) * st.st_size, hipMemcpyDeviceToHost);
    check_cuda_error("Failed to copy vector decrypted_data from device to host", err);

    start_timers(cpu_start, wc_start, ioops);
    FILE *enc_file = fopen("ciphertext_ecb", "wb+");
    FILE *dec_file = fopen("plaintext_ecb", "wb+");

    fwrite(h_encrypted_data, sizeof(BYTE) * st.st_size, 1, enc_file);
    fwrite(h_decrypted_data, sizeof(BYTE) * st.st_size, 1, dec_file);

    fclose(enc_file);
    fclose(dec_file);
    end_timers(cpu_end, wc_end, ioops);

    print_elapsed(cpu_start, wc_start, cpu_end, wc_end);

    // Free device global memory
    err = hipFree(d_data);
    check_cuda_error("Failed to free device Data", err);

    err = hipFree(d_encrypted_data);
    check_cuda_error("Failed to free device Data", err);


    err = hipFree(d_decrypted_data);
    check_cuda_error("Failed to free device Data", err);

    err = hipFree(d_key_schedule);
    check_cuda_error("Failed to free device Data", err);

    // Free host memory
    free(h_data);
    free(h_encrypted_data);
    free(h_decrypted_data);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    return TRUE;
};

int aes_cbc_test(char * path_to_file)
{
    struct timespec wc_start[STEPS_SIZE], wc_end[STEPS_SIZE];
    double cpu_start[STEPS_SIZE], cpu_end[STEPS_SIZE];
    start_timers(cpu_start, wc_start, alloc);
    BYTE *h_data;
    BYTE *d_data = NULL;
    BYTE *h_encrypted_data;
    BYTE *h_decrypted_data;
    BYTE *d_encrypted_data;
    BYTE *d_decrypted_data;
    BYTE *d_iv;
    BYTE h_iv[16] = {0x00,0x01,0x02,0x03,0x04,0x05,0x06,0x07,0x08,0x09,0x0a,0x0b,0x0c,0x0d,0x0e,0x0f};
    BYTE key[32] = {0x60,0x3d,0xeb,0x10,0x15,0xca,0x71,0xbe,0x2b,0x73,0xae,0xf0,0x85,0x7d,0x77,0x81,0x1f,0x35,0x2c,0x07,0x3b,0x61,0x08,0xd7,0x2d,0x98,0x10,0xa3,0x09,0x14,0xdf,0xf4};
    WORD h_key_schedule[60];
    WORD * d_key_schedule;
    const char *filename = path_to_file;

    //Key Setup
    aes_key_setup(key, h_key_schedule, 256);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    struct stat st;

    if (stat(filename, &st) != 0){
        printf("Could not find File");
    };

    unsigned long plaintext_size = 1;
    if(st.st_size % 16 != 0){
        plaintext_size = st.st_size + (16 - st.st_size % 16);
    } else {
        plaintext_size = st.st_size;
    }
    printf("Tamanho do Arquivo: %i \n", st.st_size);

    h_data = (BYTE *) malloc(sizeof(BYTE) * plaintext_size);

    FILE *file = fopen(filename, "rb");

    if(h_data != NULL && file){
        unsigned long current_byte = 0;

        while(fread(&h_data[current_byte], sizeof(BYTE), 1, file) == 1){
            current_byte += 1;
        };

        while(current_byte < plaintext_size - 1){
            current_byte += 1;
            h_data[current_byte] = 0;
        }
    };

    h_encrypted_data = (BYTE *) malloc(sizeof(BYTE) * plaintext_size);
    h_decrypted_data = (BYTE *) malloc(sizeof(BYTE) * plaintext_size);
    
    //Alocando o BYTE data no device
    err = hipMalloc((void **)&d_data, sizeof(BYTE) * plaintext_size);
    check_cuda_error("Failed to allocate device data", err);    

    //Alocando o BYTE d_encrypted_data no device
    err = hipMalloc((void **)&d_encrypted_data, sizeof(BYTE) * plaintext_size);
    check_cuda_error("Failed to allocate device encrypted_data", err);

    //Alocando o BYTE d_decrypted_data no device
    err = hipMalloc((void **)&d_decrypted_data, sizeof(BYTE) * plaintext_size);
    check_cuda_error("Failed to allocate device decrypted_data", err);

    //Alocando o WORD key no device
    err = hipMalloc((void **)&d_key_schedule, sizeof(WORD) * 60);
    check_cuda_error("Failed to allocate device data", err);

    //Alocando o BYTE IV no device
    err = hipMalloc((void **)&d_iv, sizeof(BYTE) * AES_BLOCK_SIZE);
    check_cuda_error("Failed to allocate device data", err);

    // Copy the host BYTE data in host memory to the device in
    // device memory
    // printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_data, h_data, sizeof(BYTE) * plaintext_size, hipMemcpyHostToDevice);
    check_cuda_error("Failed to copy BYTE data from host to device", err);

    // Copy the host WORD key_schedule in host memory to the device in
    // device memory
    // printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_key_schedule, h_key_schedule, sizeof(WORD) * 60, hipMemcpyHostToDevice);
    check_cuda_error("Failed to copy BYTE data from host to device", err);

    // Copy the host WORD key_schedule in host memory to the device in
    // device memory
    // printf("Copy input BYTE data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_iv, h_iv, sizeof(BYTE) * AES_BLOCK_SIZE, hipMemcpyHostToDevice);
    check_cuda_error("Failed to copy BYTE data from host to device", err);
    end_timers(cpu_end, wc_end, alloc);
   
    //Launch encryption
    // printf("CUDA kernel launch with 1 blocks of 1 threads\n");
    start_timers(cpu_start, wc_start, calc);
    aes_encrypt_cbc<<<1, 1>>>(d_data, plaintext_size, d_encrypted_data, d_key_schedule, 256, d_iv);
    err = hipGetLastError();
    check_cuda_error("Failed to launch aes_encrypt kernel", err);

    // Copy the device result encrypted_data in device memory to the host result vector
    // in host memory.
    // printf("Copy output encrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_encrypted_data, d_encrypted_data, sizeof(BYTE) * plaintext_size, hipMemcpyDeviceToHost);
    check_cuda_error("Failed to copy vector encrypted_data from device to host", err);


    //Launch decryption
    // printf("CUDA kernel launch with 1 blocks of 1 threads\n");
    aes_decrypt_cbc<<<1, 1>>>(d_encrypted_data, plaintext_size, d_decrypted_data, d_key_schedule, 256, d_iv);
    err = hipGetLastError();
    check_cuda_error("Failed to launch aes_decrypt kernel", err);
    hipDeviceSynchronize();
    end_timers(cpu_end, wc_end, calc);

    // Copy the device result decrypted_data in device memory to the host result vector
    // in host memory.
    // printf("Copy output decrypted_data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_decrypted_data, d_decrypted_data, sizeof(BYTE) * plaintext_size, hipMemcpyDeviceToHost);
    check_cuda_error("Failed to copy vector decrypted_data from device to host", err);

    start_timers(cpu_start, wc_start, ioops);
    FILE *enc_file = fopen("ciphertext_cbc", "wb+");
    FILE *dec_file = fopen("plaintext_cbc", "wb+");

    fwrite(h_encrypted_data, sizeof(BYTE) * plaintext_size, 1, enc_file);
    fwrite(h_decrypted_data, sizeof(BYTE) * plaintext_size, 1, dec_file);

    fclose(enc_file);
    fclose(dec_file);
    end_timers(cpu_end, wc_end, ioops);

    print_elapsed(cpu_start, wc_start, cpu_end, wc_end);


    // Free device global memory
    err = hipFree(d_data);
    check_cuda_error("Failed to free device Data", err);

    err = hipFree(d_encrypted_data);
    check_cuda_error("Failed to free device Data", err);


    err = hipFree(d_decrypted_data);
    check_cuda_error("Failed to free device Data", err);

    err = hipFree(d_iv);
    check_cuda_error("Failed to free device Data", err);

    // Free host memory
    free(h_data);
    free(h_encrypted_data);
    free(h_decrypted_data);

    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return TRUE;
};


int aes_test(char * path_to_file)
{
    int pass = 1;

    printf("AES GPU ECB Test\n");
    pass = pass && aes_ecb_test(path_to_file);
    // printf("AES GPU CBC Test\n");
    // pass = pass && aes_cbc_test(path_to_file);

    return(pass);
}

int main(int argc, char *argv[])
{
    if(argc != 2){
        printf("\nUsage:\n\t %s <relative/path/to/file>\n", argv[0]);
        return 1;
    }

    aes_test(argv[1]);
    // printf("AES Tests: %s\n", aes_test(argv[1]) ? "SUCCEEDED" : "FAILED");

    return(0);
}
